// second task
// nvcc integral.cu
// ./a.out

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void scan(const int n, float * d_in, float * d_out)
{
	int idx = threadIdx.x;
	extern __shared__ float temp[];

	int pout = 0, pin = 1;

	temp[idx] = d_in[idx];
	__syncthreads();

	for(int offset = 1; offset < n; offset = offset * 2)
	{
		pout = 1 - pout;
		pin = 1 - pout;

		if(idx>=offset)
		{
			// scan algo
			temp[pout*n+idx] = temp[pin*n+idx-offset]+temp[pin*n+idx];
		}
		else
		{
			temp[pout*n+idx]=temp[pin*n+idx];
		}
		__syncthreads();
	}
	
	d_out[idx] = temp[pout*n+idx]; 
}


int main(void)
{
	 //upper limit
        float x = 2.;

        //number of discretization points
        const int ARRAY_SIZE = pow(2, 10);
        printf("Number of discretization points: %d\n", ARRAY_SIZE);
        const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

        //filling an array with elements
        float *h_in = (float *) malloc(ARRAY_BYTES);
        float dksi = x / ARRAY_SIZE, sum=0;

        for (int i=0; i < ARRAY_SIZE; i++)
        {
                h_in[i] = exp(-pow(i*dksi, 2))*dksi;
                sum += h_in[i];
        }

        printf("Integral sequential: %.5f\n", sum);

        float *h_out = (float *) malloc(ARRAY_BYTES);
        float * d_in;
        float * d_out;

        hipMalloc(&d_in, ARRAY_BYTES);
        hipMalloc(&d_out, ARRAY_BYTES);
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	// kernel call

	scan<<<1, ARRAY_SIZE, ARRAY_BYTES*2>>>(ARRAY_SIZE, d_in, d_out);
	
	// Check for errors
        hipError_t error = hipGetLastError();
        if (error != hipSuccess)
        {
              printf("CUDA error: %s\n", hipGetErrorString(error));
              exit(-1);
        }

	hipDeviceSynchronize();
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
	
	printf("Integral cuda: %.5f\n", h_out[ARRAY_SIZE-1]);
	
	free(h_in);
	free(h_out);
	hipFree(d_in);
	hipFree(d_out);
	return 0;
}
